#include "hip/hip_runtime.h"
#include "CudaRenderer.cuh"
#include "Color.h"
#include <stdio.h>
#include <float.h>

// GLM for device code
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

// Device constants
__constant__ float EPSILON_CUDA = 1e-6f;
__constant__ int MAX_DEPTH_CUDA = 4;

// Device utility functions
__device__ bool insideTriangle(const glm::vec3& p, const glm::vec3& a, const glm::vec3& b, const glm::vec3& c) {
    return (glm::cross(b - a, p - a).z >= 0.0f &&
            glm::cross(c - b, p - b).z >= 0.0f &&
            glm::cross(a - c, p - c).z >= 0.0f);
}

__device__ glm::vec3 barycentric(const glm::vec3& p, const glm::vec3& a, const glm::vec3& b, const glm::vec3& c) {
    float area = glm::cross(b - a, c - a).z;
    if (fabs(area) < EPSILON_CUDA) return glm::vec3(0.0f);
    
    float alpha = glm::cross(b - p, c - p).z / area;
    float beta = glm::cross(c - p, a - p).z / area;
    float gamma = glm::cross(a - p, b - p).z / area;
    
    return glm::vec3(alpha, beta, gamma);
}

// 一个健壮的、可在 CUDA 设备上运行的颜色转换函数
__device__ uint32_t vec3ToColor(const glm::vec3& color) {
    // 步骤 1: Gamma 校正 (你在内核末尾已经做了，这里假设传入的已经是校正后的颜色)
    // 如果你没有在内核里做，可以在这里做

    // 步骤 2: 将颜色分量限制在 [0.0, 1.0] 的范围内，防止异常值
    float r = fmaxf(0.0f, fminf(1.0f, color.r));
    float g = fmaxf(0.0f, fminf(1.0f, color.g));
    float b = fmaxf(0.0f, fminf(1.0f, color.b));

    // 步骤 3: 将 [0.0, 1.0] 的浮点数转换为 [0, 255] 的整数
    // 乘以 255.999 是一个常用技巧，可以确保 1.0f 被正确地映射到 255
    uint8_t ri = static_cast<uint8_t>(r * 255.999f);
    uint8_t gi = static_cast<uint8_t>(g * 255.999f);
    uint8_t bi = static_cast<uint8_t>(b * 255.999f);

    // 步骤 4: 将 R, G, B 分量打包成一个 32 位整数 (uint32_t)
    // 常见的格式是 0xAARRGGBB (Alpha, Red, Green, Blue)。
    // 这里我们将 Alpha 通道设置为 255 (完全不透明)。
    return (255 << 24) | (ri << 16) | (gi << 8) | bi;
}

// 设备函数：Schlick近似法计算菲涅尔反射率
__device__ float fresnelSchlick(float cosTheta, float ior) {
    float r0 = (1.0f - ior) / (1.0f + ior);
    r0 = r0 * r0;
    // 钳制cosTheta以避免浮点数问题
    cosTheta = fmaxf(0.0f, fminf(1.0f, cosTheta));
    return r0 + (1.0f - r0) * powf(1.0f - cosTheta, 5.0f);
}

// 设备函数：计算折射方向，并处理全内反射
__device__ glm::vec3 refract_dir(const glm::vec3& I, const glm::vec3& N, float iorRatio) {
    float cosI = -glm::dot(N, I);
    float sinT2 = iorRatio * iorRatio * (1.0f - cosI * cosI);
    if (sinT2 > 1.0f) return glm::vec3(0.0f); // 全内反射发生，返回零向量
    float cosT = sqrtf(1.0f - sinT2);
    return iorRatio * I + (iorRatio * cosI - cosT) * N;
}

// Simple test kernel for CUDA functionality
__global__ void cuda_test_kernel(uint32_t* framebuffer, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Create a simple, guaranteed visible pattern
    uint32_t color;
    
    if ((x / 20 + y / 20) % 2 == 0) {
        // Red squares
        color = 0xFFFF0000;
    } else {
        // Blue squares  
        color = 0xFF0000FF;
    }
    
    framebuffer[y * width + x] = color;
}

// Rasterization kernel
__global__ void cuda_rasterize_kernel(
    uint32_t* framebuffer,
    float* zbuffer,
    int width, int height,
    const VertexShaderOutput* vertices,
    const int* indices,
    int triangleCount,
    const CudaLight* lights,
    int lightCount,
    const CudaMaterial* materials,
    const CudaCamera camera
) {
    int triangleId = blockIdx.x * blockDim.x + threadIdx.x;
    if (triangleId >= triangleCount) return;

    // Get triangle vertices
    int i0 = indices[triangleId * 3];
    int i1 = indices[triangleId * 3 + 1];
    int i2 = indices[triangleId * 3 + 2];
    
    VertexShaderOutput v0 = vertices[i0];
    VertexShaderOutput v1 = vertices[i1];
    VertexShaderOutput v2 = vertices[i2];

    // NDC to screen space
    glm::vec3 s0((v0.clipPos.x / v0.clipPos.w + 1.0f) * 0.5f * width,
                 (1.0f - v0.clipPos.y / v0.clipPos.w) * 0.5f * height,
                 (v0.clipPos.z / v0.clipPos.w + 1.0f) * 0.5f);
    glm::vec3 s1((v1.clipPos.x / v1.clipPos.w + 1.0f) * 0.5f * width,
                 (1.0f - v1.clipPos.y / v1.clipPos.w) * 0.5f * height,
                 (v1.clipPos.z / v1.clipPos.w + 1.0f) * 0.5f);
    glm::vec3 s2((v2.clipPos.x / v2.clipPos.w + 1.0f) * 0.5f * width,
                 (1.0f - v2.clipPos.y / v2.clipPos.w) * 0.5f * height,
                 (v2.clipPos.z / v2.clipPos.w + 1.0f) * 0.5f);

    // Triangle bounding box
    int minX = max(0, (int)floor(min(min(s0.x, s1.x), s2.x)));
    int maxX = min(width - 1, (int)ceil(max(max(s0.x, s1.x), s2.x)));
    int minY = max(0, (int)floor(min(min(s0.y, s1.y), s2.y)));
    int maxY = min(height - 1, (int)ceil(max(max(s0.y, s1.y), s2.y)));

    float area = glm::cross(s1 - s0, s2 - s0).z;
    if (fabs(area) < EPSILON_CUDA) return;

    // Perspective correction
    float invW0 = 1.0f / v0.w;
    float invW1 = 1.0f / v1.w;
    float invW2 = 1.0f / v2.w;

    for (int y = minY; y <= maxY; ++y) {
        for (int x = minX; x <= maxX; ++x) {
            glm::vec3 p(x + 0.5f, y + 0.5f, 0.0f);
            if (insideTriangle(p, s0, s1, s2)) {
                glm::vec3 bary = barycentric(p, s0, s1, s2);
                float z = bary.x * s0.z + bary.y * s1.z + bary.z * s2.z;
                
                int idx = y * width + x;
                if (z < zbuffer[idx]) {
                    // Atomic compare and swap for depth testing
                    float old = atomicExch(&zbuffer[idx], z);
                    if (z < old) {
                        // Interpolate attributes
                        float invW = bary.x * invW0 + bary.y * invW1 + bary.z * invW2;
                        glm::vec3 worldPos = (v0.worldPos * bary.x * invW0 + 
                                            v1.worldPos * bary.y * invW1 + 
                                            v2.worldPos * bary.z * invW2) / invW;
                        glm::vec3 normal = glm::normalize(v0.normal * bary.x * invW0 + 
                                                        v1.normal * bary.y * invW1 + 
                                                        v2.normal * bary.z * invW2);
                        glm::vec2 uv = (v0.uv * bary.x * invW0 + 
                                       v1.uv * bary.y * invW1 + 
                                       v2.uv * bary.z * invW2) / invW;

                        // Simple Phong shading
                        glm::vec3 color(0.1f); // ambient
                        if (lightCount > 0) {
                            glm::vec3 viewDir = glm::normalize(camera.position - worldPos);
                            for (int i = 0; i < lightCount; ++i) {
                                glm::vec3 lightDir = glm::normalize(lights[i].position - worldPos);
                                float diff = fmaxf(0.0f, glm::dot(normal, lightDir));
                                glm::vec3 reflectDir = glm::normalize(2.0f * glm::dot(normal, lightDir) * normal - lightDir);
                                float spec = powf(fmaxf(0.0f, glm::dot(reflectDir, viewDir)), 16.0f);
                                
                                color += (diff + spec) * lights[i].color * lights[i].intensity;
                            }
                        }
                        
                        framebuffer[idx] = vec3ToColor(color);
                    } else {
                        zbuffer[idx] = old; // restore original depth
                    }
                }
            }
        }
    }
}

// Ray tracing kernels
__device__ bool intersectTriangle(const glm::vec3& rayOrigin, const glm::vec3& rayDir,
                                const CudaTriangle& tri, float& t, glm::vec2& uv) {
    glm::vec3 edge1 = tri.v1 - tri.v0;
    glm::vec3 edge2 = tri.v2 - tri.v0;
    glm::vec3 h = glm::cross(rayDir, edge2);
    float a = glm::dot(edge1, h);
    
    if (a > -EPSILON_CUDA && a < EPSILON_CUDA) return false;
    
    float f = 1.0f / a;
    glm::vec3 s = rayOrigin - tri.v0;
    float u = f * glm::dot(s, h);
    
    if (u < 0.0f || u > 1.0f) return false;
    
    glm::vec3 q = glm::cross(s, edge1);
    float v = f * glm::dot(rayDir, q);
    
    if (v < 0.0f || u + v > 1.0f) return false;
    
    t = f * glm::dot(edge2, q);
    uv = glm::vec2(u, v);
    
    return t > EPSILON_CUDA;
}

// Create orthonormal basis from normal vector
// This function is used to create a tangent and bitangent for normal mapping
__device__ void createOrthonormalBasis(const glm::vec3& n, glm::vec3& t, glm::vec3& b) {
    if (fabsf(n.x) > fabsf(n.z)) {
        t = glm::vec3(-n.y, n.x, 0.0f);
    } else {
        t = glm::vec3(0.0f, -n.z, n.y);
    }
    t = glm::normalize(t);
    b = glm::cross(n, t);
}

__global__ void cuda_raytrace_kernel(
    uint32_t* framebuffer,
    int width, int height,
    const CudaTriangle* triangles,
    int triangleCount,
    const CudaMaterial* materials,
    const CudaLight* lights,
    int lightCount,
    const CudaCamera camera,
    int samplesPerPixel,
    int maxDepth
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    // 初始化随机状态
    hiprandState state;
    hiprand_init((unsigned long long)y * width + x, 0, 0, &state);

    glm::vec3 color(0.0f);

    if (triangleCount == 0) {
        float u = (float)x / width;
        float v = (float)y / height;
        color = glm::vec3(u, v, 0.5f);
        framebuffer[y * width + x] = vec3ToColor(color);
        return;
    }

    // 每个像素多次采样以实现抗锯齿
    for (int s = 0; s < samplesPerPixel; ++s) {
        float px = (float)x + hiprand_uniform(&state);
        float py = (float)y + hiprand_uniform(&state);

        // 生成光线
        float u = (px / width) * 2.0f - 1.0f;
        float v = (py / height) * 2.0f - 1.0f;
        v = -v; // 翻转y轴

        float fovRad = camera.fov * 3.14159265359f / 180.0f;
        float scale = tanf(fovRad * 0.5f);
        u *= scale * camera.aspect;
        v *= scale;

        glm::vec3 rayDir = glm::normalize(camera.forward + u * camera.right + v * camera.up);
        
        if (isnan(rayDir.x) || isnan(rayDir.y) || isnan(rayDir.z)) {
            color += glm::vec3(1.0f, 0.0f, 0.0f);
            continue;
        }

        // 追踪光线
        glm::vec3 rayOrigin = camera.position;
        glm::vec3 sampleColor(0.0f);
        glm::vec3 throughput(1.0f);

        for (int depth = 0; depth < maxDepth; ++depth) {
            float closestT = FLT_MAX;
            int hitTriangle = -1;
            glm::vec2 hitUV;

            // 寻找最近的交点
            for (int i = 0; i < triangleCount; ++i) {
                float t;
                glm::vec2 uv;
                if (intersectTriangle(rayOrigin, rayDir, triangles[i], t, uv)) {
                    if (t < closestT) {
                        closestT = t;
                        hitTriangle = i;
                        hitUV = uv;
                    }
                }
            }

            if (hitTriangle == -1) {
                sampleColor += throughput * glm::vec3(0.3f, 0.5f, 0.8f);
                break;
            }

            glm::vec3 hitPoint = rayOrigin + closestT * rayDir;
            const CudaTriangle& tri = triangles[hitTriangle];
            const CudaMaterial& mat = materials[tri.materialId];

            float w = 1.0f - hitUV.x - hitUV.y;
            glm::vec3 shadingNormal = w * tri.n0 + hitUV.x * tri.n1 + hitUV.y * tri.n2;

            if (glm::dot(shadingNormal, shadingNormal) < 1e-6f) {
                shadingNormal = glm::cross(tri.v1 - tri.v0, tri.v2 - tri.v0);
                if (glm::dot(shadingNormal, shadingNormal) < 1e-6f) {
                    sampleColor += throughput * glm::vec3(1.0f, 0.0f, 1.0f);
                    break;
                }
            }
            shadingNormal = glm::normalize(shadingNormal);
            // glm::vec3 frontNormal = glm::faceforward(shadingNormal, -rayDir, shadingNormal);
            glm::vec3 frontNormal = shadingNormal; // 假设法线总是朝向外部

            // --- 【核心修改】统一的、基于物理的材质模型 ---

            // 1. 为不透明物体添加直接光照贡献 (Next Event Estimation)
            if (mat.transparency < 0.5f) {
                glm::vec3 lightContrib(0.0f); 
                for (int i = 0; i < lightCount; ++i) {
                    glm::vec3 toLight = lights[i].position - hitPoint;
                    float lightDistSq = glm::dot(toLight, toLight);
                    if (lightDistSq < 1e-6f) continue;
                    float lightDist = sqrtf(lightDistSq);
                    glm::vec3 lightDir = toLight / lightDist;

                    // --- 【核心修正】阴影光线检测 ---
                    bool inShadow = false;
                    glm::vec3 shadowRayOrigin = hitPoint + frontNormal * 1e-4f; // 加上一个小的偏移量防止自相交
                    for (int j = 0; j < triangleCount; ++j) {
                        float t_shadow;
                        glm::vec2 uv_shadow;
                        if (intersectTriangle(shadowRayOrigin, lightDir, triangles[j], t_shadow, uv_shadow)) {
                            // 如果交点在物体表面和光源之间，则说明该点处于阴影中
                            if (t_shadow > 0.0f && t_shadow < lightDist) {
                                inShadow = true;
                                break; // 找到遮挡物，无需继续检查
                            }
                        }
                    }

                    // 如果不在阴影中，才添加该光源的贡献
                    if (!inShadow) {
                        float NdotL = fmaxf(0.0f, glm::dot(frontNormal, lightDir));
                        // 这里可以根据距离添加衰减: lightContrib += (NdotL * lights[i].color * lights[i].intensity) / lightDistSq;
                        lightContrib += NdotL * lights[i].color * lights[i].intensity;
                    }
                }
                // 只有漫反射部分接收直接光
                glm::vec3 diffuseColor = (1.0f - mat.metallic) * mat.baseColor;
                sampleColor += throughput * diffuseColor * lightContrib;
            }

            // --- 【优化 1】俄式轮盘赌 (Russian Roulette) ---
            // 在几次弹射后，根据光线能量决定是否继续追踪
            if (depth > 3) {
                float p = fmaxf(throughput.x, fmaxf(throughput.y, throughput.z));
                if (hiprand_uniform(&state) > p) {
                    break; // 概率性地终止路径
                }
                throughput /= p; // 补偿能量，保持无偏
            }

            // 2. 根据材质类型，决定光线的下一次弹射行为
            
            // --- 情况一: 透明材质 (如玻璃) ---
            if (mat.transparency > 0.5f) {
                glm::vec3 outward_normal;
                float iorRatio;
                float cosTheta = glm::dot(rayDir, frontNormal);
                float reflectance;

                if (cosTheta < 0.0f) { // 光线从外部射入物体
                    outward_normal = frontNormal;
                    iorRatio = 1.0f / mat.ior;
                    cosTheta = -cosTheta;
                } else { // 光线从物体内部射出
                    outward_normal = -frontNormal;
                    iorRatio = mat.ior;
                }

                glm::vec3 refractedDir = refract_dir(rayDir, outward_normal, iorRatio);
                if (glm::dot(refractedDir, refractedDir) < 1e-6f) { // 发生全内反射
                    reflectance = 1.0f;
                } else {
                    reflectance = fresnelSchlick(cosTheta, mat.ior);
                }
                
                if (hiprand_uniform(&state) < reflectance) { // 概率上选择反射
                    rayDir = glm::reflect(rayDir, frontNormal);
                    rayOrigin = hitPoint + frontNormal * 1e-4f;
                } else { // 概率上选择折射
                    rayDir = glm::normalize(refractedDir);
                    rayOrigin = hitPoint - frontNormal * 1e-4f; // 光线进入物体内部
                    throughput *= mat.baseColor; // 对于有色玻璃，光能被基色过滤
                }
            } 
            // --- 情况二: 不透明材质 ---
            else {
                // 子情况 2a: 电介质 (非金属)
                if (mat.metallic < 0.5f) {
                    float reflectance = fresnelSchlick(-glm::dot(rayDir, frontNormal), mat.ior);
                    if (hiprand_uniform(&state) < reflectance) { // 镜面反射
                        float roughness = fmaxf(mat.roughness, 0.01f);
                        float alpha = roughness * roughness;
                        float r1 = hiprand_uniform(&state);
                        float r2 = hiprand_uniform(&state);
                        float theta = acosf(sqrtf((1.0f - r1) / (1.0f + (alpha * alpha - 1.0f) * r1)));
                        float phi = 2.0f * 3.14159265359f * r2;
                        glm::vec3 h_local(sinf(theta) * cosf(phi), sinf(theta) * sinf(phi), cosf(theta));
                        glm::vec3 tangent, bitangent;
                        createOrthonormalBasis(frontNormal, tangent, bitangent);
                        glm::vec3 h_world = glm::normalize(h_local.x * tangent + h_local.y * bitangent + h_local.z * frontNormal);
                        rayDir = glm::reflect(rayDir, h_world);
                        rayOrigin = hitPoint + frontNormal * 1e-4f;
                        // 电介质的镜面反射是白色的，所以 throughput 不变
                    } else { // 漫反射
                        glm::vec3 tangent, bitangent;
                        createOrthonormalBasis(frontNormal, tangent, bitangent);
                        float r1 = hiprand_uniform(&state);
                        float r2 = hiprand_uniform(&state);
                        float sinTheta = sqrtf(r1);
                        float cosTheta = sqrtf(1.0f - r1);
                        float phi = 2.0f * 3.14159265359f * r2;
                        glm::vec3 localDir(sinTheta * cosf(phi), sinTheta * sinf(phi), cosTheta);
                        rayDir = glm::normalize(localDir.x * tangent + localDir.y * bitangent + localDir.z * frontNormal);
                        rayOrigin = hitPoint + frontNormal * 1e-4f;
                        throughput *= mat.baseColor; // 漫反射的光能被基色过滤
                    }
                }
                // 子情况 2b: 金属
                else {
                    // 金属总是镜面反射
                    float roughness = fmaxf(mat.roughness, 0.01f);
                    float alpha = roughness * roughness;
                    float r1 = hiprand_uniform(&state);
                    float r2 = hiprand_uniform(&state);
                    float theta = acosf(sqrtf((1.0f - r1) / (1.0f + (alpha * alpha - 1.0f) * r1)));
                    float phi = 2.0f * 3.14159265359f * r2;
                    glm::vec3 h_local(sinf(theta) * cosf(phi), sinf(theta) * sinf(phi), cosf(theta));
                    glm::vec3 tangent, bitangent;
                    createOrthonormalBasis(frontNormal, tangent, bitangent);
                    glm::vec3 h_world = glm::normalize(h_local.x * tangent + h_local.y * bitangent + h_local.z * frontNormal);
                    rayDir = glm::reflect(rayDir, h_world);
                    rayOrigin = hitPoint + frontNormal * 1e-4f;
                    throughput *= mat.baseColor; // 金属的反射是有颜色的
                }
            }
        }
        // --- 【优化 2】样本辉度钳制 (Sample Clamping) ---
        // 限制单次光路追踪的贡献，有效消除火花/亮斑
        if (!isnan(sampleColor.x) && !isnan(sampleColor.y) && !isnan(sampleColor.z)) {
            sampleColor = glm::min(sampleColor, glm::vec3(4.0f)); // 将单次采样的最大亮度限制在4.0
        } else {
            sampleColor = glm::vec3(0.0f); // 丢弃无效采样
        }
        color += sampleColor;
    }

    color /= float(samplesPerPixel);

    // Gamma 校正
    color.r = powf(fmaxf(color.r, 0.0f), 1.0f / 2.2f);
    color.g = powf(fmaxf(color.g, 0.0f), 1.0f / 2.2f);
    color.b = powf(fmaxf(color.b, 0.0f), 1.0f / 2.2f);

    framebuffer[y * width + x] = vec3ToColor(color);
}

__global__ void cuda_clear_kernel(uint32_t* framebuffer, float* zbuffer, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        framebuffer[idx] = 0xFF000000;
        zbuffer[idx] = FLT_MAX;
    }
}

// Host functions
extern "C" {
    void cuda_render_rasterize(
        uint32_t* framebuffer,
        float* zbuffer,
        int width, int height,
        const VertexShaderOutput* vertices,
        const int* indices,
        int triangleCount,
        const CudaLight* lights,
        int lightCount,
        const CudaMaterial* materials,
        const CudaCamera& camera
    ) {
        dim3 blockSize(256);
        dim3 gridSize((triangleCount + blockSize.x - 1) / blockSize.x);
        
        cuda_rasterize_kernel<<<gridSize, blockSize>>>(
            framebuffer, zbuffer, width, height,
            vertices, indices, triangleCount,
            lights, lightCount, materials, camera
        );
        
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void cuda_render_raytrace(
        uint32_t* framebuffer,
        int width, int height,
        const CudaTriangle* triangles,
        int triangleCount,
        const CudaMaterial* materials,
        const CudaLight* lights,
        int lightCount,
        const CudaCamera& camera,
        int samplesPerPixel,
        int maxDepth
    ) {
        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                      (height + blockSize.y - 1) / blockSize.y);
        
        cuda_raytrace_kernel<<<gridSize, blockSize>>>(
            framebuffer, width, height,
            triangles, triangleCount, materials,
            lights, lightCount, camera,
            samplesPerPixel, maxDepth
        );
        
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void cuda_clear_buffers(uint32_t* framebuffer, float* zbuffer, int size) {
        dim3 blockSize(256);
        dim3 gridSize((size + blockSize.x - 1) / blockSize.x);
        
        cuda_clear_kernel<<<gridSize, blockSize>>>(framebuffer, zbuffer, size);
        
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void cuda_test_pattern(uint32_t* framebuffer, int width, int height) {
        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                      (height + blockSize.y - 1) / blockSize.y);
        
        cuda_test_kernel<<<gridSize, blockSize>>>(framebuffer, width, height);
        
        CUDA_CHECK(hipDeviceSynchronize());
    }
}

// CudaRenderer class implementation
CudaRenderer::CudaRenderer() : initialized(false), d_framebuffer(nullptr), d_zbuffer(nullptr),
    d_triangles(nullptr), d_materials(nullptr), d_lights(nullptr),
    d_vertices(nullptr), d_indices(nullptr) {
}

CudaRenderer::~CudaRenderer() {
    cleanup();
}

void CudaRenderer::initialize(int width, int height, int maxTriangles, int maxMaterials, int maxLights) {
    if (initialized) cleanup();
    
    framebuffer_size = width * height;
    max_triangles = maxTriangles;
    max_materials = maxMaterials;
    max_lights = maxLights;
    max_vertices = maxTriangles * 3;
    max_indices = maxTriangles * 3;
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_framebuffer, framebuffer_size * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_zbuffer, framebuffer_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_triangles, max_triangles * sizeof(CudaTriangle)));
    CUDA_CHECK(hipMalloc(&d_materials, max_materials * sizeof(CudaMaterial)));
    CUDA_CHECK(hipMalloc(&d_lights, max_lights * sizeof(CudaLight)));
    CUDA_CHECK(hipMalloc(&d_vertices, max_vertices * sizeof(VertexShaderOutput)));
    CUDA_CHECK(hipMalloc(&d_indices, max_indices * sizeof(int)));
    
    initialized = true;
}

void CudaRenderer::cleanup() {
    if (!initialized) return;
    
    hipFree(d_framebuffer);
    hipFree(d_zbuffer);
    hipFree(d_triangles);
    hipFree(d_materials);
    hipFree(d_lights);
    hipFree(d_vertices);
    hipFree(d_indices);
    
    initialized = false;
}

void CudaRenderer::renderRaytracing(
    uint32_t* framebuffer,
    int width, int height,
    const CudaTriangle* triangles,
    int triangleCount,
    const CudaMaterial* materials,
    int materialCount,
    const CudaLight* lights,
    int lightCount,
    const CudaCamera& camera,
    int samplesPerPixel,
    int maxDepth
) {
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_triangles, triangles, triangleCount * sizeof(CudaTriangle), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_materials, materials, materialCount * sizeof(CudaMaterial), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lights, lights, lightCount * sizeof(CudaLight), hipMemcpyHostToDevice));
    
    // Call CUDA kernel
    cuda_render_raytrace(
        d_framebuffer, width, height,
        d_triangles, triangleCount, d_materials,
        d_lights, lightCount, camera,
        samplesPerPixel, maxDepth
    );
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(framebuffer, d_framebuffer, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

void CudaRenderer::renderRasterization(
    uint32_t* framebuffer,
    float* zbuffer,
    int width, int height,
    const VertexShaderOutput* vertices,
    const int* indices,
    int triangleCount,
    const CudaLight* lights,
    int lightCount,
    const CudaMaterial* materials,
    const CudaCamera& camera
) {
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_vertices, vertices, triangleCount * 3 * sizeof(VertexShaderOutput), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_indices, indices, triangleCount * 3 * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_materials, materials, triangleCount * sizeof(CudaMaterial), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lights, lights, lightCount * sizeof(CudaLight), hipMemcpyHostToDevice));
    
    // Call CUDA kernel
    cuda_render_rasterize(
        d_framebuffer, d_zbuffer, width, height,
        d_vertices, d_indices, triangleCount,
        d_lights, lightCount, d_materials, camera
    );
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(framebuffer, d_framebuffer, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(zbuffer, d_zbuffer, width * height * sizeof(float), hipMemcpyDeviceToHost));
}

void CudaRenderer::clearBuffers(uint32_t* framebuffer, float* zbuffer, int size) {
    cuda_clear_buffers(d_framebuffer, d_zbuffer, size);
    // Copy cleared buffers back to host if needed
    CUDA_CHECK(hipMemcpy(framebuffer, d_framebuffer, size * sizeof(uint32_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(zbuffer, d_zbuffer, size * sizeof(float), hipMemcpyDeviceToHost));
}

void CudaRenderer::renderTestPattern(uint32_t* framebuffer, int width, int height) {
    printf("CudaRenderer::renderTestPattern called with %dx%d\n", width, height);
    
    // Call CUDA test kernel
    cuda_test_pattern(d_framebuffer, width, height);
    
    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(error));
        return;
    }
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(framebuffer, d_framebuffer, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost));
    
    // Check first few pixels to verify data
    printf("First few pixels: %08X %08X %08X %08X\n", 
           framebuffer[0], framebuffer[1], framebuffer[2], framebuffer[3]);
}
