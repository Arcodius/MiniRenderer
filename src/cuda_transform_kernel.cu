
#include <hip/hip_runtime.h>
// // File: cuda_transform_kernel.cu

// #include <cuda_runtime.h>
// #include <device_launch_parameters.h>

// struct float3 { float x, y, z; };
// struct float2 { float x, y; };
// struct float4 { float x, y, z, w; };

// __device__ float4 make_float4(float3 v, float w) {
//     return { v.x, v.y, v.z, w };
// }

// __device__ float dot(float4 a, float4 b) {
//     return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
// }

// // Column-major 4x4 matrix
// struct float4x4 {
//     float m[4][4]; // m[column][row]
// };

// __device__ float4 mul(const float4x4& mat, const float4& vec) {
//     float4 res;
//     res.x = mat.m[0][0] * vec.x + mat.m[1][0] * vec.y + mat.m[2][0] * vec.z + mat.m[3][0] * vec.w;
//     res.y = mat.m[0][1] * vec.x + mat.m[1][1] * vec.y + mat.m[2][1] * vec.z + mat.m[3][1] * vec.w;
//     res.z = mat.m[0][2] * vec.x + mat.m[1][2] * vec.y + mat.m[2][2] * vec.z + mat.m[3][2] * vec.w;
//     res.w = mat.m[0][3] * vec.x + mat.m[1][3] * vec.y + mat.m[2][3] * vec.z + mat.m[3][3] * vec.w;
//     return res;
// }

// struct Vertex {
//     float3 worldPos;
//     float3 normal;
//     float2 uv;
// };

// struct ClippedVertex {
//     Vertex vertex;
//     float4 clipPos;
// };

// __global__ void transformKernel(
//     const Vertex* vertices,
//     const unsigned int* indices,
//     float4x4 mvp,
//     ClippedVertex* output,
//     int triangleCount
// ) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= triangleCount) return;

//     int i = idx * 3;
//     unsigned int idx0 = indices[i];
//     unsigned int idx1 = indices[i + 1];
//     unsigned int idx2 = indices[i + 2];

//     Vertex v0 = vertices[idx0];
//     Vertex v1 = vertices[idx1];
//     Vertex v2 = vertices[idx2];

//     float4 pos0 = make_float4(v0.worldPos, 1.0f);
//     float4 pos1 = make_float4(v1.worldPos, 1.0f);
//     float4 pos2 = make_float4(v2.worldPos, 1.0f);

//     output[i + 0] = { v0, mul(mvp, pos0) };
//     output[i + 1] = { v1, mul(mvp, pos1) };
//     output[i + 2] = { v2, mul(mvp, pos2) };
// }
